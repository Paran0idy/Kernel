#include "hip/hip_runtime.h"
#include "kernel.h"

#define OFFSET(i, j, N) (i) * (N) + (j)
__global__ void naive_kernel(float *a, float *b, float *c, int M, int N, int K) {
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    int tx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ty < M && tx < N) {
        for (int i = 0; i < K; i++)
            c[OFFSET(ty, tx, N)] += a[OFFSET(ty, i, K)] * b[OFFSET(i, tx, N)];
    }
}

#define OFFSET(i, j, N) (i) * (N) + (j)
#define FLOAT4(pointer) reinterpret_cast<float4*> (&pointer)[0]
__global__ void threadTiling_kernel(float *a, float *b ,float *c, int M, int N, int K) {
    const int BM = 128, BN = 128, BK = 8, TILE = 8;
    int row = blockIdx.y * BM;
    int col = blockIdx.x * BN;

    __shared__ float sa[BM][BK], sb[BK][BN];
    float result[TILE][TILE] = {0};

    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    int smem_a_m = tid / 2;
    int smem_a_k = (tid % 2) << 2;

    int smem_b_k = tid / 32;
    int smem_b_n = (tid % 32) << 2;

    int gmem_a_m = row + smem_a_m;
    int gmem_b_n = col + smem_b_n;

    for (int k = 0; k < K / BK; k++) {
        int gmem_a_k = k * BK;
        int gmem_b_k = k * BK;

        // G2S
        FLOAT4(sa[smem_a_m][smem_a_k]) = FLOAT4(a[OFFSET(gmem_a_m, gmem_a_k, K)]);
        FLOAT4(sb[smem_b_k][smem_b_n]) = FLOAT4(b[OFFSET(gmem_b_k, gmem_b_n, N)]);
        __syncthreads();

        // compute
        int ty = threadIdx.y * TILE;
        int tx = threadIdx.x * TILE;
        for (int kk = 0; kk < BK; kk++) 
            for (int i = 0; i < TILE; i++)
                for (int j = 0; j < TILE; j++)
                    result[i][j] += sa[ty + i][kk] * sb[kk][tx + j];
        __syncthreads();
    }

    // S2G
    int ty = row + threadIdx.y * TILE;
    int tx = col + threadIdx.x * TILE;
    for (int i = 0; i < TILE; i++)
        for (int j = 0; j < TILE; j += 4)
            FLOAT4(c[OFFSET(ty + i, tx + j, N)]) = FLOAT4(result[i][j]);
}
void naive() {
    const int M = 32, N = 32, K = 32;
    const int BM = 32, BN = 32, BK = 32;

    size_t size_a = M * K * sizeof(float);
    size_t size_b = K * N * sizeof(float);
    size_t size_c = M * N * sizeof(float);

    float *a = (float *)malloc(size_a);
    float *b = (float *)malloc(size_b);
    float *c = (float *)malloc(size_c);

    for (int i = 0; i < M * K; i++) a[i] = 1.0;
    for (int i = 0; i < K * N; i++) b[i] = 1.0;

    float *da, *db, *dc;
    hipMalloc(&da, size_a);
    hipMalloc(&db, size_b);
    hipMalloc(&dc, size_c);

    hipMemcpy(da, a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(db, b, size_b, hipMemcpyHostToDevice);

    dim3 block(N / BN, M / BM);
    dim3 thread(BN, BM);
    naive_kernel<<<block, thread>>>(da, db, dc, M, N, K);

    hipMemcpy(c, dc, size_c, hipMemcpyDeviceToHost);

    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++)
            std::cout << c[OFFSET(i, j, N)] << " ";
        std::cout << std::endl;
    }

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    free(a);
    free(b);
    free(c);
}

void threadTiling() {
    const int M = 128, N = 128, K = 128;
    const int BM = 128, BN = 128, BK = 8, TILE = 8;

    size_t size_a = M * K * sizeof(float);
    size_t size_b = K * N * sizeof(float);
    size_t size_c = M * N * sizeof(float);

    float *a = (float *)malloc(size_a);
    float *b = (float *)malloc(size_b);
    float *c = (float *)malloc(size_c);

    for (int i = 0; i < M * K; i++) a[i] = 1.0;
    for (int i = 0; i < K * N; i++) b[i] = 1.0;

    float *da, *db, *dc;
    hipMalloc(&da, size_a);
    hipMalloc(&db, size_b);
    hipMalloc(&dc, size_c);

    hipMemcpy(da, a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(db, b, size_b, hipMemcpyHostToDevice);

    dim3 block(N / BN, M / BM);
    dim3 thread(BN / TILE, BM / TILE);
    threadTiling_kernel<<<block, thread>>>(da, db, dc, M, N, K);

    hipMemcpy(c, dc, size_c, hipMemcpyDeviceToHost);

    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++)
            std::cout << c[OFFSET(i, j, N)] << " ";
        std::cout << std::endl;
    }

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    free(a);
    free(b);
    free(c);
}





int main() {
    naive();
    threadTiling();

    return 0;
}